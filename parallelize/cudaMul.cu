#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <algorithm>
#include <math.h>
#include <sys/timeb.h>
#include <time.h>
#include "timing.h"

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
// #include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
// #include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

// Includes, kernels
// #include "simpleAtomicIntrinsics_kernel.cuh"


using namespace std;

__global__
void kernel_init_zero(double *res, int k) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= k)
		return;

	res[index] = 0;
}

__global__
void kernel_init_zero_2(double *res, double *res2, int k) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= k)
		return;

	res[index] = 0;
	res2[index] = 0;
}

__global__
void kernel_calc(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	//int end = index+for_loop_size;
	//for(int p=index;p<end;p++) {
	int p = index;
	int ix = i[p];
	int jx = j[p];
	double xx = x[p];

	// res[jx] += xx*v[ix];
	atomicAdd(&res[jx], xx*v[ix]);


	// res[ix] += xx*v[jx];
    atomicAdd(&res[ix], xx*v[jx]);		
	//}
}



__global__
void kernel_calc_2(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if(index >= m)
                return;

	int ix,jx;
	double xx;
	int p = index;
    int i1 = ix = i[p];
    int j1 = jx = j[p];
    double x1 = xx = x[p];
    int i2 = ix = i[p+1];
    int j2 = jx = j[p+1];
    double x2 = xx = x[p+1];
	
    ix = i1;
    jx = j1;
    xx = x1;
    atomicAdd(&res[ix], xx*v[jx]);
    ix = i2;
    jx = j2;
    xx = x2;
	atomicAdd(&res[ix], xx*v[jx]);


	ix = i1;
	jx = j1;
	xx = x1;
    atomicAdd(&res[jx], xx*v[ix]);
    ix = i2;
    jx = j2;
    xx = x2;
    atomicAdd(&res[jx], xx*v[ix]);
}



__global__
void kernel_calc_2for(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= m)
        return;

    int end = index+for_loop_size;
    for(int p=index;p<end;p++) {
        int ix = i[p];
        int jx = j[p];
        double xx = x[p];

        // res[ix] += xx*v[jx];
        atomicAdd(&res[ix], xx*v[jx]);
    }
    for(int p=index;p<end;p++) {
        int ix = i[p];
        int jx = j[p];
        double xx = x[p];

        // res[jx] += xx*v[ix];
        atomicAdd(&res[jx], xx*v[ix]);
	}	
}


__global__
void kernel_calc_res_ix(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	int end = index+for_loop_size;
	for(int p=index;p<end;p++) {
		int ix = i[p];
		int jx = j[p];
		double xx = x[p];

		// res[ix] += xx*v[jx];
		atomicAdd(&res[ix], xx*v[jx]);

	}
}


__global__
void kernel_calc_res_jx(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	int end = index+for_loop_size;
	for(int p=index;p<end;p++) {
		int ix = i[p];
		int jx = j[p];
		double xx = x[p];

		// res[jx] += xx*v[ix];
		atomicAdd(&res[jx], xx*v[ix]);
	}
}







__global__
void kernel_calc_with_shared(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	int end = index+for_loop_size;
	__shared__ int shared_i[512];
	__shared__ int shared_j[512];
	for(int p=index;p<end;p++) {
		shared_i[threadIdx.x+p-index] = i[index];
		shared_j[threadIdx.x+p-index] = j[index];
	}

	__syncthreads();

	for(int p=index;p<end;p++) {
		int ix = shared_i[threadIdx.x+p-index];
		int jx = shared_j[threadIdx.x+p-index];
		double xx = x[p];

		// res[ix] += xx*v[jx];
		atomicAdd(&res[ix], xx*v[jx]);

		// res[jx] += xx*v[ix];
		atomicAdd(&res[jx], xx*v[ix]);
	}
}



__global__
void kernel_calc_with_shared_v(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	int end = index+for_loop_size;

	__shared__ double shared_v_i[1024];
	__shared__ double shared_v_j[1024];
	// __shared__ double shared_res[512];
	for(int p=index;p<end;p++) {
		int index_internal = threadIdx.x+p-index;
		shared_v_i[index_internal] = v[i[p]];
		shared_v_j[index_internal] = v[j[p]];
		// shared_res[index_internal] = 0;
	}

	__syncthreads();

	for(int p=index;p<end;p++) {
		int index_internal = threadIdx.x+p-index;
		double xx = x[p];

		// res[ix] += xx*v[jx];
		atomicAdd(&res[i[p]], xx*shared_v_j[index_internal]);

		// res[jx] += xx*v[ix];
		atomicAdd(&res[j[p]], xx*shared_v_i[index_internal]);
	}
}




__global__
void kernel_calc_output_shared(int k, int m, int *i, int *j, double *x, double *v, double *res, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	int index2 = index * for_loop_size;

	for(int p=0;p<m;p++) {
		int ix = i[p];
		int jx = j[p];

		if(ix >= index2 && ix < index2+for_loop_size) {
			double xx = x[p];
			res[ix] += xx*v[jx];
			// atomicAdd(&res[ix], xx*v[jx]);
		}

		if(jx >= index2 && jx < index2+for_loop_size) {
			double xx = x[p];
			res[jx] += xx*v[ix];
			// atomicAdd(&res[jx], xx*v[ix]);
		}
	}
}





__global__
void kernel_calc_sep_mem(int k, int m, int *i, int *j, double *x, double *v, double *res, double *res2, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= m)
		return;

	int end = index+for_loop_size;
	for(int p=index;p<end;p++) {
		int ix = i[p];
		int jx = j[p];
		double xx = x[p];

		// res[ix] += xx*v[jx];
		atomicAdd(&res[ix], xx*v[jx]);

		// res2[jx] += xx*v[ix];
		// atomicAdd(&res2[jx], xx*v[ix]);

	}
}

__global__
void kernel_calc_summation(int k, int m, double *res, double *res2, int for_loop_size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= k)
		return;

	int end = index+for_loop_size;
	for(int p=index;p<end;p++) {
		res[p] += res2[p];
	}
}







void cudaMul(int *i,int *j,double *x,int m,double *v,int k,double *res, int max_ij, int thread_count, int Q, double simpleTime) {
	int *d_i,*d_j;
	double *d_x,*d_v,*d_res,*d_res2;

	hipMalloc((void**) &d_i, sizeof(int) * m);
	hipMemcpy(d_i, i, sizeof(int)*m, hipMemcpyDefault);
	hipMalloc((void**) &d_j, sizeof(int) * m);
	hipMemcpy(d_j, j, sizeof(int)*m, hipMemcpyDefault);
	hipMalloc((void**) &d_x, sizeof(double) * m);
	hipMemcpy(d_x, x, sizeof(double)*m, hipMemcpyDefault);
	hipMalloc((void**) &d_v, sizeof(double) * k);
	hipMemcpy(d_v, v, sizeof(double)*k, hipMemcpyDefault);

	hipMalloc((void**) &d_res, sizeof(double) * k);
	hipMalloc((void**) &d_res2, sizeof(double) * k);

	int for_loop_size, delta;

	hipStream_t st1, st2;
	hipStreamCreate(&st1);
	hipStreamCreate(&st2);

	// struct timeb tm0,tm1;
	// ftime(&tm0);
	timingBegin();
	for(int q=0;q<Q;q++) {
		// int p;
		// #pragma omp parallel for private(p)
		// for (p=0;p<k;p++) 
		// 	res[p] = 0;
		int vector = 1024;
		int gang = (k+vector-1) / vector;
		kernel_init_zero<<<gang, vector>>>(d_res, k);
		// kernel_init_zero<<<gang, vector>>>(d_res2, k);
		// kernel_init_zero_2<<<gang, vector>>>(d_res, d_res2, k);


		for_loop_size = 1;
		vector = 64;
		delta = for_loop_size * vector;
		gang = (m+delta-1) / delta;

		kernel_calc<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);
                
		// kernel_calc_2<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);

		//kernel_calc_2for<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);

		// delta should be less than 512
		// kernel_calc_with_shared<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);

		// kernel_calc_output_shared<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);


//		kernel_calc_res_ix<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);
//		kernel_calc_res_jx<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);

		// kernel_calc_with_shared_v<<<gang, vector>>>(k, m, d_i, d_j, d_x, d_v, d_res, for_loop_size);


		// kernel_calc_sep_mem<<<gang, vector, 0, st1>>>(k, m, d_i, d_j, d_x, d_v, d_res, d_res2, for_loop_size);
		// kernel_calc_sep_mem<<<gang, vector, 0, st2>>>(k, m, d_j, d_i, d_x, d_v, d_res2, d_res2, for_loop_size);
		// for_loop_size = 1;
		// vector = 1024;
		// delta = for_loop_size * vector;
		// gang = (k+delta-1) / delta;
		// hipStreamSynchronize(st1);
		// hipStreamSynchronize(st2);
		// kernel_calc_summation<<<gang, vector, 0, st1>>>(k, m, d_res, d_res2, for_loop_size);
		// hipStreamSynchronize(st1);

		hipDeviceSynchronize();
	}
	// ftime(&tm1);
	timingEnd();

	// hipStreamSynchronize(st1);
	// hipStreamSynchronize(st2);
	hipStreamDestroy(st1);
	hipStreamDestroy(st2);

	if(simpleTime > 0) { 
		// double timing = (tm1.time - tm0.time)*1000.0 + (tm1.millitm - tm0.millitm);
		double timing = timingCount();
		printf("CUDA multiplication took %lg milliseconds\n", timing/Q);
		printf("speedup is %.2f\n", simpleTime/timing);
	}

	hipMemcpy(res, d_res, sizeof(double)*k, hipMemcpyDefault);

	hipFree(d_i);
	hipFree(d_j);
	hipFree(d_x);
	hipFree(d_v);
	hipFree(d_res);
	if(d_res2)
		hipFree(d_res2);

}

