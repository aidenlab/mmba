
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



#define UTMV_DEFAULT_VECTOR_SIZE 1024


template<class T>
__global__
void gpu_kernel_res_init(int k, T *res) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= k)
		return;

	res[index] = 0;
}


__device__ 
double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__
void gpu_kernel_utmv(int *m, int **i, int **j, double **x, double *v, double *res) {
	int ic = blockIdx.x;
	int *ii = i[ic];
	int *jj = j[ic];
	double *xx = x[ic];
	int mic = m[ic];

	int iter_count = (mic + blockDim.x - 1) / blockDim.x;

	// int index2 = blockIdx.x * blockDim.x + threadIdx.x;
	// if(index2 == 0)
	// 	printf("blockIdx.x = %d - iter_count = %d\n", blockDim.x, iter_count);


	int index = threadIdx.x;
	for(int p=0;p < iter_count;p++) {

		// int index = threadIdx.x + p*blockDim.x;
		index += blockDim.x;

		if(index >= mic)
			break;

		int ix = ii[index];
		int jx = jj[index];
		double xv = xx[index];

		// res[jx] += xx*v[ix];
		atomicAdd(&res[jx], xv*v[ix]);

		// res[ix] += xx*v[jx];
		atomicAdd(&res[ix], xv*v[jx]);
	}
}


__global__
void gpu_kernel_utmv_one_kernel_per_ic(int *m, int **i, int **j, double **x, double *v, double *res, int ic) {
	int *ii = i[ic];
	int *jj = j[ic];
	double *xx = x[ic];
	int mic = m[ic];

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= mic)
		return;

	int ix = ii[index];
	int jx = jj[index];
	double xv = xx[index];

	// res[jx] += xx*v[ix];
	atomicAddDouble(&res[jx], xv*v[ix]);

	// res[ix] += xx*v[jx];
	atomicAddDouble(&res[ix], xv*v[jx]);
}


static int first_time_visit = 0;
static int **d_i = NULL;
static int **d_j = NULL;
static double **d_x = NULL;
static int *d_m = NULL;
static double *d_v = NULL;
static double *d_res = NULL;


template<class T>
inline
void mmbaCudaAllocMatrix(T **dev, int c, int *m) {
	T **temp = (T**) malloc(sizeof(T*) * c);
	for(int p=0;p<c;p++) {
		T *q;
		if(hipMalloc((void**) &q, sizeof(T) * m[p]) != hipSuccess) {
			printf("Unable to allocate memory!\n");
			exit(1);
		}
		temp[p] = q;
	}

	hipMemcpy(dev, temp, sizeof(T*) * c, hipMemcpyDefault);
	free(temp);
}


template<class T>
inline
void mmbaCudaMemcpy(T **to, T **from, int c, int *m) {
	T **temp = (T**) malloc(sizeof(T*) * c);
	hipMemcpy(temp, to, sizeof(T*) * c, hipMemcpyDefault);
	for(int i=0;i<c;i++) {
		hipMemcpy(temp[i], from[i], sizeof(T) * m[i], hipMemcpyDefault);
	}
	free(temp);
}


inline
void utmvCudaAlloc(int c, int **i, int **j, double **x, int *m, double *v, int k, double *res) {
	if(first_time_visit)
		return;

	first_time_visit = 1;

	if(hipMalloc((void***) &d_i, sizeof(int*) * c) != hipSuccess) {
		printf("Unable to allocate memory on line %d!\n", __LINE__);
		exit(1);
	}
	mmbaCudaAllocMatrix<int>(d_i, c, m);
	mmbaCudaMemcpy<int>(d_i, i, c, m);

	hipMalloc((void***) &d_j, sizeof(int*) * c);
	mmbaCudaAllocMatrix<int>(d_j, c, m);
	mmbaCudaMemcpy<int>(d_j, j, c, m);

	hipMalloc((void***) &d_x, sizeof(double*) * c);
	mmbaCudaAllocMatrix<double>(d_x, c, m);
	mmbaCudaMemcpy<double>(d_x, x, c, m);



	hipMalloc((void**) &d_m, sizeof(int) * c);
	hipMemcpy(d_m, m, sizeof(int) * c, hipMemcpyDefault);

	hipMalloc((void**) &d_v, sizeof(double) * k);

	hipMalloc((void**) &d_res, sizeof(double) * k);

}


// #define ST_COUNT 8

void utmvMulCuda(int c, int **i, int **j, double **x, int *m, double *v, int k, double *res) {
	int vector, gang;

	utmvCudaAlloc(c, i, j, x, m, v, k, res);

	hipMemcpy(d_v, v, sizeof(double) * k, hipMemcpyDefault);

	vector = UTMV_DEFAULT_VECTOR_SIZE;
	gang = (k+vector-1) / vector;
	gpu_kernel_res_init<double><<<gang, vector>>>(k, d_res);
	// cudaDeviceSynchronize();


/*
 * Considering all ic'es at once
 */
	vector = UTMV_DEFAULT_VECTOR_SIZE;
	gang = c;
	gpu_kernel_utmv<<<gang, vector>>>(d_m, d_i, d_j, d_x, d_v, d_res);


/*
 * Running a kernel per ic
 */
#if 0
	vector = 1024;
	gang = (m[0] + vector - 1) / vector;
	cudaStream_t st[ST_COUNT];
	for(int st_i=0;st_i < ST_COUNT;st_i++)
		cudaStreamCreate(&st[st_i]);
	for(int ic = 0;ic < c;ic+=ST_COUNT) {
		for(int qc=0;qc < ST_COUNT;qc++)
			gpu_kernel_utmv_one_kernel_per_ic<<<gang, vector, 0, st[qc]>>>(d_m, d_i, d_j, d_x, d_v, d_res, ST_COUNT*ic + qc);
	}
	// cudaDeviceSynchronize();
#endif

	hipMemcpy(res, d_res, sizeof(double) * k, hipMemcpyDefault);
}

#ifndef BUILD_PYTHON_LIB
void utmvMul(int c, int **i, int **j, double **x, int *m, double *v, int k, double *res) {
	utmvMulCuda(c, i, j, x, m, v, k, res);
}
#endif
